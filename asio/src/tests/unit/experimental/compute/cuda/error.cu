#include "asio/experimental/compute/cuda/error.hpp"
#include <cassert>

int main()
{
  std::error_code e1 = hipErrorInvalidValue;
  assert(e1 == hipErrorInvalidValue);
  assert(e1.message() == "invalid argument");
}
